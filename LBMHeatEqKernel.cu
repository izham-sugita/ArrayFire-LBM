#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include "blocksize.h"

//play with the block size to find your best performance
//hint: blockDim_x should be bigger than blockDim_y
//hint: blockDim_x*blockDim_y should not exceed 1024 (gpu dependent)
// #define  blockDim_x       32
// #define  blockDim_y       32


/*fused advection-collision*/
__global__ void fusedAdvColl
(
int nx,
int ny,
float omega,
float* f1,
float* f2,
float* f3,
float* f4,
float* feq,
float* fn1,
float* fn2,
float* fn3,
float* fn4
)
{
   int    j,    jx,   jy;
   jy = blockDim.y*blockIdx.y + threadIdx.y;
   jx = blockDim.x*blockIdx.x + threadIdx.x;

if(jx>0 && jx<nx-1){
if(jy>0 && jy<ny-1){   

/*Slow memory access*/
/* 
j = ny*jx + jy;
fn1[j+ny] = (1.0 - omega)*f1[j] + omega*feq[j]; 
fn2[j+1] = (1.0 - omega)*f2[j] + omega*feq[j]; 
fn3[j-ny] = (1.0 - omega)*f3[j] + omega*feq[j]; 
fn4[j-1] = (1.0 - omega)*f4[j] + omega*feq[j];
*/

/*
j = nx*jy + jx;
fn1[j+1] = (1.0 - omega)*f1[j] + omega*feq[j]; 
fn2[j+nx] = (1.0 - omega)*f2[j] + omega*feq[j]; 
fn3[j-1] = (1.0 - omega)*f3[j] + omega*feq[j]; 
fn4[j-nx] = (1.0 - omega)*f4[j] + omega*feq[j];
*/

j = nx*jy + jx;
fn1[j] = (1.0 - omega)*f1[j-1] + omega*feq[j-1];
fn2[j] = (1.0 - omega)*f2[j-nx] + omega*feq[j-nx];
fn3[j] = (1.0 - omega)*f3[j+1] + omega*feq[j+1];
fn4[j] = (1.0 - omega)*f4[j+nx] + omega*feq[j+nx];

  }
 }


}

__global__ void macroscopic2
(
int nx,
int ny,
float* fn1,
float* fn2,
float* fn3,
float* fn4,
float* f1,
float* f2,
float* f3,
float* f4,
float* Tempn,
float* feq
)
{
   int    j,    jx,   jy;

   jy = blockDim.y*blockIdx.y + threadIdx.y;
   jx = blockDim.x*blockIdx.x + threadIdx.x;

   //j = ny*jx + jy; //cause slow memory access
   j = nx*jy + jx;

	Tempn[j] = fn1[j] + fn2[j] + fn3[j] + fn4[j];
	feq[j] = 0.25*Tempn[j];
	f1[j] = fn1[j];
	f2[j] = fn2[j];
	f3[j] = fn3[j];
	f4[j] = fn4[j];

}

/*--fused all doesn't work prefectly.. why? */
__global__ void fusedAll
(
int nx,
int ny,
float omega,
float* f1,
float* f2,
float* f3,
float* f4,
float* feq,
float* fn1,
float* fn2,
float* fn3,
float* fn4,
float* Tempn
)
{

   int    j,    jx,   jy;
   jy = blockDim.y*blockIdx.y + threadIdx.y;
   jx = blockDim.x*blockIdx.x + threadIdx.x;

if(jx>0 && jx<nx-1){
if(jy>0 && jy<ny-1){   

j = nx*jy + jx;
fn1[j] = (1.0 - omega)*f1[j-1] + omega*feq[j-1]; 
fn2[j] = (1.0 - omega)*f2[j-nx] + omega*feq[j-nx]; 
fn3[j] = (1.0 - omega)*f3[j+1] + omega*feq[j+1]; 
fn4[j] = (1.0 - omega)*f4[j+nx] + omega*feq[j+nx];
Tempn[j] = fn1[j] + fn2[j] + fn3[j] + fn4[j];

}}

__syncthreads();

/* Updating */
if(jx>0 && jx<nx-1){
if(jy>0 && jy<ny-1){
feq[j] = 0.25*Tempn[j];
f1[j] = fn1[j];
f2[j] = fn2[j];
f3[j] = fn3[j];
f4[j] = fn4[j];

}
}

}


//kernel wrapper
float  LBMdiffusion2d
// ====================================================================
//
// purpose    :  2-dimensional diffusion equation solved by LBM
//
// date       :  July 9, 2018
// programmer :  Muhammad Izham aka Sugita
// place      :  Universiti Malaysia Perlis
//
(
   int      nx,         /* x-dimensional grid size                   */
   int      ny,         /* y-dimensional grid size                   */
   float    omega,      /* relaxation parameter                      */
float* f1,
float* f2,
float* f3, 
float* f4,
float* fn1,
float* fn2,
float* fn3,
float* fn4,
float* feq,
float* Tempn
)
{

dim3  grid(nx/blockDim_x,ny/blockDim_y,1);
dim3  threads(blockDim_x,blockDim_y,1);

/*
collision<<<grid,threads>>>(nx,ny,omega,f1,f2,f3,f4, feq);
advection<<<grid,threads>>>(nx,ny,f1,f2,f3,f4,fn1,fn2,fn3,fn4);
macroscopic<<<grid,threads>>>(nx,ny,fn1,fn2,fn3,fn4,Tempn,feq);
*/

/*
fusedAdvColl<<<grid,threads>>>(nx,ny,omega,f1,f2,f3,f4,feq,fn1,fn2,fn3,fn4);
macroscopic2<<<grid,threads>>>(nx,ny,fn1,fn2,fn3,fn4,f1,f2,f3,f4,Tempn,feq);
*/

/*Fixed!*/
fusedAll<<<grid,threads>>>(nx,ny,omega,f1,f2,f3,f4,feq,fn1,fn2,fn3,fn4,Tempn);

return (float)(nx*ny)*7.0;

}


